#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <cnpy.h>

/*
float test_spmv_csr(hipsparseHandle_t handle,
                    hipsparseMatDescr_t descr,
                    int* cu_csrRowPtr,
                    int* cu_csrColInd,
                    float* cu_csrVal,
                    int num_rows,
                    int num_cols,
                    int nnz,
                    int num_runs) {
    // seems dense matrix (InVec and OutVec) is treated as column-major in hipsparseScsrmm2; check again

    float alpha = 1.0;
    float beta = 0.0;

    // warm up run
    hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        num_rows, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
        cu_InVec, &beta, cu_OutVec);
    hipDeviceSynchronize();

    // measure time
    float elapsed_time_ms = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            num_rows, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
            cu_InVec, &beta, cu_OutVec);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    // hipMemcpy(OutVec, cu_OutVec, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < 8; i++) {
    //     std::cout << OutVec[i] << std::endl;
    // }

    // clean up
    free(InVec);
    free(OutVec);
    hipFree(cu_InVec);
    hipFree(cu_OutVec);

    return elapsed_time_ms / num_runs / 1000;
}
*/

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int benchmark_spmv_csr(std::string dataset) {
    // load csr matrix
    cnpy::npz_t npz = cnpy::npz_load(dataset);
    cnpy::NpyArray npy_shape = npz["shape"];
    cnpy::NpyArray npy_data = npz["data"];
    cnpy::NpyArray npy_indptr = npz["indptr"];
    cnpy::NpyArray npy_indices = npz["indices"];
    int nnz = npy_data.shape[0];
    int num_rows = npy_shape.data<int>()[0];
    int num_cols = npy_shape.data<int>()[2];
    float* csrVal = npy_data.data<float>();
    int* csrRowPtr = npy_indptr.data<int>();
    int* csrColInd = npy_indices.data<int>();
    float alpha = 1.0;
    float beta = 0.0;

    // std::cout << "nnz:" << nnz << std::endl;
    // std::cout << "num_rows:" << num_rows << std::endl;
    // std::cout << "num_cols:" << num_cols << std::endl;

    hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4, cudaStat5;

    // device malloc
    float* cu_csrVal=0;
    cudaStat1 = hipMalloc((void**)&cu_csrVal, nnz * sizeof(float));
    int* cu_csrRowPtr=0;
    cudaStat2 = hipMalloc((void**)&cu_csrRowPtr, (num_rows + 1) * sizeof(int));
    int* cu_csrColInd=0;
    cudaStat3 = hipMalloc((void**)&cu_csrColInd, nnz * sizeof(int));
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat1 = hipMemcpy(cu_csrVal, csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cu_csrRowPtr, csrRowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cu_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    float* InVec = (float*)malloc(sizeof(float) * num_cols);
    for (int i = 0; i < num_cols; i++) {
        InVec[i] = 1.0;
    }
    float* OutVec = (float*)malloc(sizeof(float) * num_rows);
    for (int i = 0; i < num_rows; i++) {
        OutVec[i] = 0.0;
    }

    // device malloc
    float* cu_InVec=0;
    cudaStat4 = hipMalloc((void**)&cu_InVec, num_cols * sizeof(float));
    float* cu_OutVec=0;
    cudaStat5 = hipMalloc((void**)&cu_OutVec, num_rows * sizeof(float));
    if ((cudaStat1 != hipSuccess) || (cudaStat2 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat4 = hipMemcpy(cu_InVec, InVec, num_cols * sizeof(float), hipMemcpyHostToDevice);
    cudaStat5 = hipMemcpy(cu_OutVec, OutVec, num_rows * sizeof(float), hipMemcpyHostToDevice);
    if ((cudaStat4 != hipSuccess) || (cudaStat5 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    // initialize cusparse library
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, num_rows, num_cols, nnz,
                                      cu_csrRowPtr, cu_csrColInd, cu_csrVal,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector input
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, num_cols, cu_InVec, HIP_R_32F) )
    // Create dense vector output
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, num_rows, cu_OutVec, HIP_R_32F) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
    hipDeviceSynchronize();

    int num_runs = VAR;
    float elapsed_time_ms = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                     HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    std::cout << "Total time = " << elapsed_time_ms / 1000 << " s" << std::endl;
    std::cout << "average_time = " << elapsed_time_ms / num_runs << " ms" << std::endl;
    std::cout << "nnz is " << nnz << std::endl;
    double throughput = double(nnz) * double(2 * num_runs) / double(elapsed_time_ms) / 1000 / 1000;
    std::cout << "THROUGHPUT = " << throughput << " GOPS" << std::endl;

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    // free memory
    hipFree(cu_csrVal);
    hipFree(cu_csrColInd);
    hipFree(cu_csrRowPtr);
    hipFree(cu_InVec);
    hipFree(cu_OutVec);


    return 0;
}


int main(int argc, char** argv) {
    hipSetDevice(6);
    std::string dataset = argv[1];
    benchmark_spmv_csr(dataset);
}
