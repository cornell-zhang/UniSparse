#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <cnpy.h>


int test_spmm_csr(hipsparseHandle_t handle,
                  hipsparseMatDescr_t descr,
                  int* cu_csrRowPtr,
                  int* cu_csrColInd,
                  float* cu_csrVal,
                  int num_rows,
                  int num_cols,
                  int nnz,
                  int feat_len,
                  int num_runs) {
    // seems dense matrix (FeatMatrix and OutMatrix) is treated as column-major in hipsparseScsrmm2; check again
    float* FeatMatrix = (float*)malloc(sizeof(float) * num_cols * feat_len);
    for (int i = 0; i < num_cols * feat_len; i++) {
        FeatMatrix[i] = 1.0;
    }
    float* OutMatrix = (float*)malloc(sizeof(float) * num_cols * feat_len);
    for (int i = 0; i < num_cols * feat_len; i++) {
        OutMatrix[i] = 2.0;
    }

    hipError_t cudaStat1, cudaStat2;

    // device malloc
    float* cu_FeatMatrix=0;
    cudaStat1 = hipMalloc((void**)&cu_FeatMatrix, num_cols * feat_len * sizeof(float));
    float* cu_OutMatrix=0;
    cudaStat2 = hipMalloc((void**)&cu_OutMatrix, num_rows * feat_len * sizeof(float));
    if ((cudaStat1 != hipSuccess) || (cudaStat2 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat1 = hipMemcpy(cu_FeatMatrix, FeatMatrix, num_cols * feat_len * sizeof(float), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cu_OutMatrix, OutMatrix, num_rows * feat_len * sizeof(float), hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) || (cudaStat2 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    float alpha = 1.0;
    float beta = 0.0;

    hipsparseStatus_t status;

    // warm up run
    status= hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        num_rows, feat_len, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
        cu_FeatMatrix, feat_len, &beta, cu_OutMatrix, num_rows);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("cusparseSPMM failed\n");
        exit(-1);
    }
    hipDeviceSynchronize();

    // measure time
    float elapsed_time = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        status= hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            num_rows, feat_len, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
            cu_FeatMatrix, feat_len, &beta, cu_OutMatrix, num_rows);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    std::cout << "average time of " << num_runs << " runs: " << elapsed_time / num_runs << " ms" << std::endl;

    // hipMemcpy(OutMatrix, cu_OutMatrix, num_nodes * feat_len * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < 8; i++) {
    //     std::cout << OutMatrix[i] << std::endl;
    // }

    // clean up
    free(FeatMatrix);
    free(OutMatrix);
    hipFree(cu_FeatMatrix);
    hipFree(cu_OutMatrix);

    return 0;
}


int benchmark_spmm_csr() {
    std::string file_name = "/work/shared/users/phd/yh457/data/sparse_matrix_graph/uniform_100K_100_csr_float32.npz";

    // load csr matrix
    cnpy::npz_t npz = cnpy::npz_load(file_name);
    cnpy::NpyArray npy_shape = npz["shape"];
    cnpy::NpyArray npy_data = npz["data"];
    cnpy::NpyArray npy_indptr = npz["indptr"];
    cnpy::NpyArray npy_indices = npz["indices"];
    int nnz = npy_data.shape[0];
    int num_rows = npy_shape.data<int>()[0];
    int num_cols = npy_shape.data<int>()[2];
    float* csrVal = npy_data.data<float>();
    int* csrRowPtr = npy_indptr.data<int>();
    int* csrColInd = npy_indices.data<int>();

    // std::cout << "nnz:" << nnz << std::endl;
    // std::cout << "num_rows:" << num_rows << std::endl;
    // std::cout << "num_cols:" << num_cols << std::endl;

    hipError_t cudaStat1, cudaStat2, cudaStat3;

    // device malloc
    float* cu_csrVal=0;
    cudaStat1 = hipMalloc((void**)&cu_csrVal, nnz * sizeof(float));
    int* cu_csrRowPtr=0;
    cudaStat2 = hipMalloc((void**)&cu_csrRowPtr, (num_rows + 1) * sizeof(int));
    int* cu_csrColInd=0;
    cudaStat3 = hipMalloc((void**)&cu_csrColInd, nnz * sizeof(int));
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat1 = hipMemcpy(cu_csrVal, csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cu_csrRowPtr, csrRowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cu_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    hipsparseStatus_t status;

    // initialize cusparse library
    hipsparseHandle_t handle=0;
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE Library initialization failed\n");
        exit(-1);
    }

    // create and setup matrix descriptor
    hipsparseMatDescr_t descr=0;
    status= hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("Matrix descriptor initialization failed\n");
        exit(-1);
    }
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    std::vector<int> feat_len_values{1, 4, 32, 64, 128, 256, 512};
    int num_runs = 10;
    for (int feat_len : feat_len_values) {
        std::cout << "\nfeat_len is: " << feat_len << std::endl;
        test_spmm_csr(handle, descr, cu_csrRowPtr, cu_csrColInd, cu_csrVal, num_rows, num_cols, nnz, feat_len, num_runs);
    }

    // destroy handle
    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE Library release of resources failed\n");
        exit(-1);
    }

    // free memory
    hipFree(cu_csrVal);
    hipFree(cu_csrColInd);
    hipFree(cu_csrRowPtr);

    return 0;
}


int main () {
    benchmark_spmm_csr();
}
