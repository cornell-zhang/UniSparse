#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <cnpy.h>

/*
float test_spmv_csr(hipsparseHandle_t handle,
                    hipsparseMatDescr_t descr,
                    int* cu_csrRowPtr,
                    int* cu_csrColInd,
                    float* cu_csrVal,
                    int num_rows,
                    int num_cols,
                    int nnz,
                    int num_runs) {
    // seems dense matrix (InVec and OutVec) is treated as column-major in hipsparseScsrmm2; check again

    float alpha = 1.0;
    float beta = 0.0;

    // warm up run
    hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        num_rows, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
        cu_InVec, &beta, cu_OutVec);
    hipDeviceSynchronize();

    // measure time
    float elapsed_time_ms = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            num_rows, num_cols, nnz, &alpha, descr, cu_csrVal, cu_csrRowPtr, cu_csrColInd,
            cu_InVec, &beta, cu_OutVec);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    // hipMemcpy(OutVec, cu_OutVec, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < 8; i++) {
    //     std::cout << OutVec[i] << std::endl;
    // }

    // clean up
    free(InVec);
    free(OutVec);
    hipFree(cu_InVec);
    hipFree(cu_OutVec);

    return elapsed_time_ms / num_runs / 1000;
}
*/

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int benchmark_spmv_csr(std::string dataset) {
    // load csr matrix
    cnpy::npz_t npz = cnpy::npz_load(dataset);
    cnpy::NpyArray npy_shape = npz["shape"];
    cnpy::NpyArray npy_data = npz["data"];
    cnpy::NpyArray npy_indptr = npz["indptr"];
    cnpy::NpyArray npy_indices = npz["indices"];
    int nnz = npy_data.shape[0];
    int num_rows = npy_shape.data<int>()[0];
    int num_cols = npy_shape.data<int>()[2];
    int num_cols_b = 40;
    float* csrVal = npy_data.data<float>();
    int* csrRowPtr = npy_indptr.data<int>();
    int* csrColInd = npy_indices.data<int>();
    float alpha = 1.0;
    float beta = 0.0;

    // std::cout << "nnz:" << nnz << std::endl;
    // std::cout << "num_rows:" << num_rows << std::endl;
    // std::cout << "num_cols:" << num_cols << std::endl;

    hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4, cudaStat5;

    // device malloc
    float* cu_csrVal=0;
    cudaStat1 = hipMalloc((void**)&cu_csrVal, nnz * sizeof(float));
    int* cu_csrRowPtr=0;
    cudaStat2 = hipMalloc((void**)&cu_csrRowPtr, (num_rows + 1) * sizeof(int));
    int* cu_csrColInd=0;
    cudaStat3 = hipMalloc((void**)&cu_csrColInd, nnz * sizeof(int));
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat1 = hipMemcpy(cu_csrVal, csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cu_csrRowPtr, csrRowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cu_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) ||
        (cudaStat3 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    float* InMat = (float*)malloc(sizeof(float) * num_cols * num_cols_b);
    for (int i = 0; i < num_cols * num_cols_b; i++) {
        InMat[i] = 1.0;
    }
    float* OutMat = (float*)malloc(sizeof(float) * num_rows * num_cols_b);
    for (int i = 0; i < num_rows * num_cols_b; i++) {
        OutMat[i] = 0.0;
    }

    // device malloc
    float* cu_InMat=0;
    cudaStat4 = hipMalloc((void**)&cu_InMat, num_cols * num_cols_b * sizeof(float));
    float* cu_OutMat=0;
    cudaStat5 = hipMalloc((void**)&cu_OutMat, num_rows * num_cols_b * sizeof(float));
    if ((cudaStat1 != hipSuccess) || (cudaStat2 != hipSuccess)) {
        printf("Device malloc failed");
        exit(-1);
    }

    // memcpy from host to device
    cudaStat4 = hipMemcpy(cu_InMat, InMat, num_cols * num_cols_b * sizeof(float), hipMemcpyHostToDevice);
    cudaStat5 = hipMemcpy(cu_OutMat, OutMat, num_rows * num_cols_b * sizeof(float), hipMemcpyHostToDevice);
    if ((cudaStat4 != hipSuccess) || (cudaStat5 != hipSuccess)) {
        printf("Memcpy from Host to Device failed");
        exit(-1);
    }

    // initialize cusparse library
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matX, matY;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, num_rows, num_cols, nnz,
                                      cu_csrRowPtr, cu_csrColInd, cu_csrVal,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector input
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matX, num_cols, num_cols_b, num_cols, cu_InMat, HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense vector output
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matY, num_rows, num_cols_b, num_rows, cu_OutMat, HIP_R_32F, HIPSPARSE_ORDER_COL) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
				 HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                 &alpha, matA, matX, &beta, matY, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matX, &beta, matY, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    hipDeviceSynchronize();

    int num_runs = VAR;
    float elapsed_time_ms = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, matX, &beta, matY, HIP_R_32F,
                                     HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    std::cout << "Total time = " << elapsed_time_ms / 1000 << "s" << std::endl;
    std::cout << "average_time = " << elapsed_time_ms / num_runs << " ms" << std::endl;
    std::cout << "nnz is " << nnz << " and num_cols is " << num_cols << std::endl;
    double throughput = double(nnz) * double(2 * num_cols_b * num_runs) / double(elapsed_time_ms) / 1000 / 1000;
    std::cout << "THROUGHPUT = " << throughput << " GOPS" << std::endl;

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matX) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    // free memory
    hipFree(cu_csrVal);
    hipFree(cu_csrColInd);
    hipFree(cu_csrRowPtr);
    hipFree(cu_InMat);
    hipFree(cu_OutMat);


    return 0;
}


int main(int argc, char** argv) {
    hipSetDevice(0);
    std::string dataset = argv[1];
    benchmark_spmv_csr(dataset);
}
